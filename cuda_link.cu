#include "hip/hip_runtime.h"
#include <stdio.h>
#include <complex>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

extern "C" { 
#include "cuda_link.h" 
    }


 __global__ void gpu_scattering ( int i, int N, int total_sc, double flu_part, double *f0, int *atyp, int jj, double qvect1, double qvect2, double qvect3, double (*cq)[3], double (*CELL)[3], double scellx, double scelly, double scellz, hipDoubleComplex *dev_dummy_array_1, hipDoubleComplex *dev_dummy_array_2 )
 {
    int ii = threadIdx.x + blockIdx.x * blockDim.x;
    double test_exp;        
    test_exp = qvect1 * (cq[ii][0]  + CELL[i][0] * scellx) + qvect2 * (cq[ii][1]  + CELL[i][1] * scelly) + qvect3 * (cq[ii][2]  + CELL[i][2] * scellz)  ; 
    dev_dummy_array_1[ii] = make_hipDoubleComplex( 0.0 , test_exp );
 }

extern "C"
int cuda_function( int i ,
                    int N ,
                    int total_sc ,
                    double flu_part ,
                    int lcount ,
                    int rcount,
                    double *f0,
                    int *atyp,
                    int jj,
                    double qvect1,
                    double qvect2,
                    double qvect3,
                    double (*cq)[3],
                    double (*CELL)[3],
                    double scellx,
                    double scelly,
                    double scellz,
                    double *dummy_array_1,
                    double  *dummy_array_2 )
   {
    
   int  *dev_atyp;
   double *dev_f0;
   double *dev_cq[3];
   double *dev_CELL[3];
   hipDoubleComplex *dev_dummy_array_1;
   hipDoubleComplex *dev_dummy_array_2;
    

    hipMalloc( (void**)& dev_f0 , rcount * lcount * sizeof (double) );
    hipMalloc( (void**)& dev_CELL , total_sc * 3 * sizeof (double) );
    hipMalloc( (void**)& dev_cq , N * 3 * sizeof (double) );
    hipMalloc( (void**)& dev_atyp ,N * sizeof (int) );
    hipMalloc( (void**)& dev_dummy_array_1 , N * sizeof(hipDoubleComplex)  ); 
    hipMalloc( (void**)& dev_dummy_array_2 , N * sizeof(hipDoubleComplex)  ); 
   
   
    hipMemcpyAsync( dev_f0, f0, rcount * lcount * sizeof(double), hipMemcpyHostToDevice );
    hipMemcpyAsync( dev_CELL, CELL , total_sc * 3 * sizeof(double), hipMemcpyHostToDevice );
    hipMemcpyAsync( dev_cq, cq , N * 3 * sizeof(double), hipMemcpyHostToDevice );
    hipMemcpyAsync( dev_atyp, atyp , N * sizeof(int), hipMemcpyHostToDevice );
//    hipMemcpyAsync( dev_dummy_array_1, dummy_array_1, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice );
  //  hipMemcpyAsync( dev_dummy_array_2, dummy_array_2, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice );

 gpu_scattering<<<1, 10>>>( i, N, total_sc, flu_part, dev_f0, dev_atyp, jj, 
                              qvect1,  qvect2,  qvect3, cq, CELL, scellx, 
                              scelly, scellz, dev_dummy_array_1, dev_dummy_array_2 ) ;//best: constant number of blocks + threads (64+128)

   return 0;
    
    }
